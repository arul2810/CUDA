﻿/*

File Name : HelloWorld.cu

File Description: Print Hello World using CPU and CPU

File Author : Arul Prakash Samathuvamani - hello@arulprakash.dev - https://arulprakash.dev

Date: 6/10/2021

*/


// Declare Headers

#include <stdio.h>
#include "hip/hip_runtime.h"


// Declaration of function to run on CPU
void cpu_run_function() {

	printf("This Function is run by the CPU \n"); // This runs on CPU

}

// Declaration of function that is to be run on GPU.
__global__ 
void gpu_run_function() {

	printf("This function is run by GPU \n"); // This runs on GPU

}

// Main function, runs on CPU

int main() {
	 
	printf("Running the function \n");


	cpu_run_function(); // Call CPU function

	gpu_run_function <<<10, 10 >>> (); // Call GPU function with 10 blocks each with 10 threads

	hipDeviceSynchronize(); // Wait for the process to finish

	cpu_run_function(); // Run CPU function again

	// Replicate this function without cudaDeviceSynchronise()

	return 0;
}