﻿#include <stdio.h>
#include "hip/hip_runtime.h"



void cpu_run_function() {

	printf("This Function is run by the CPU \n");

}

__global__ 
void gpu_run_function() {

	printf("This function is run by GPU \n");

}


int main() {

	printf("Running the function \n");


	cpu_run_function();

	gpu_run_function <<<10, 10 >>> ();

	hipDeviceSynchronize();

	cpu_run_function();

	return 0;
}